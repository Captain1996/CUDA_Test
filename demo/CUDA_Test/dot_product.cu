#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <algorithm>
#include <memory>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

/* __global__: 函数类型限定符;在设备上运行;在主机端调用,计算能力3.2及以上可以在
设备端调用;声明的函数的返回值必须是void类型;对此类型函数的调用是异步的,即在
设备完全完成它的运行之前就返回了;对此类型函数的调用必须指定执行配置,即用于在
设备上执行函数时的grid和block的维度,以及相关的流(即插入<<<   >>>运算符);
a kernel,表示此函数为内核函数(运行在GPU上的CUDA并行计算函数称为kernel(内核函
数),内核函数必须通过__global__函数类型限定符定义);*/
__global__ static void dot_product(const float* A, const float* B, float* partial_C, int elements_num)
{
	/* __shared__: 变量类型限定符；使用__shared__限定符，或者与__device__限
	定符连用，此时声明的变量位于block中的共享存储器空间中，与block具有相同
	的生命周期，仅可通过block内的所有线程访问；__shared__和__constant__变量
	默认为是静态存储；在__shared__前可以加extern关键字，但表示的是变量大小
	由执行参数确定；__shared__变量在声明时不能初始化；可以将CUDA C的关键字
	__shared__添加到变量声明中，这将使这个变量驻留在共享内存中；CUDA C编译
	器对共享内存中的变量与普通变量将分别采取不同的处理方式 */
	__shared__ float cache[256]; // == threadsPerBlock

	/* gridDim: 内置变量,用于描述线程网格的维度,对于所有线程块来说,这个
	变量是一个常数,用来保存线程格每一维的大小,即每个线程格中线程块的数量.
	为dim3类型；
	blockDim: 内置变量,用于说明每个block的维度与尺寸.为dim3类型,包含
	了block在三个维度上的尺寸信息;对于所有线程块来说,这个变量是一个常数,
	保存的是线程块中每一维的线程数量;
	blockIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程块的索引;用
	于说明当前thread所在的block在整个grid中的位置,blockIdx.x取值范围是
	[0,gridDim.x-1],blockIdx.y取值范围是[0, gridDim.y-1].为uint3类型,
	包含了一个block在grid中各个维度上的索引信息;
	threadIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程索引;用于
	说明当前thread在block中的位置;如果线程是一维的可获取threadIdx.x,如果
	是二维的还可获取threadIdx.y,如果是三维的还可获取threadIdx.z;为uint3类
	型,包含了一个thread在block中各个维度的索引信息 */
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float tmp{ 0.f };
	while (tid < elements_num) {
		tmp += A[tid] * B[tid];
		tid += blockDim.x * gridDim.x;
	}

	// 设置cache中相应位置上的值
	// 共享内存缓存中的偏移就等于线程索引；线程块索引与这个偏移无关，因为每
	// 个线程块都拥有该共享内存的私有副本
	cache[cacheIndex] = tmp;

	/* __syncthreads: 对线程块中的线程进行同步；CUDA架构将确保，除非线程块
	中的每个线程都执行了__syncthreads()，否则没有任何线程能执行
	__syncthreads()之后的指令;在同一个block中的线程通过共享存储器(shared 
	memory)交换数据，并通过栅栏同步(可以在kernel函数中需要同步的位置调用
	__syncthreads()函数)保证线程间能够正确地共享数据；使用clock()函数计时，
	在内核函数中要测量的一段代码的开始和结束的位置分别调用一次clock()函数，
	并将结果记录下来。由于调用__syncthreads()函数后，一个block中的所有
	thread需要的时间是相同的，因此只需要记录每个block执行需要的时间就行了，
	而不需要记录每个thread的时间 */
	__syncthreads();

	// 对于规约运算来说，以下code要求threadPerBlock必须是2的指数
	int i = blockDim.x / 2;
	while (i != 0) {
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];

		// 在循环迭代中更新了共享内存变量cache，并且在循环的下一次迭代开始之前，
		// 需要确保当前迭代中所有线程的更新操作都已经完成
		__syncthreads();
		i /= 2;
	}

	// 只有cacheIndex == 0的线程执行这个保存操作，这是因为只有一个值写入到
	// 全局内存，因此只需要一个线程来执行这个操作，当然你也可以选择任何一个
	// 线程将cache[0]写入到全局内存
	if (cacheIndex == 0)
		partial_C[blockIdx.x] = cache[0];
}

static int dot_product_gpu_1(const float* A, const float* B, float* value, int elements_num, float* elapsed_time)
{
	/* hipEvent_t: CUDA event types,结构体类型, CUDA事件,用于测量GPU在某
	个任务上花费的时间,CUDA中的事件本质上是一个GPU时间戳,由于CUDA事件是在
	GPU上实现的,因此它们不适于对同时包含设备代码和主机代码的混合代码计时*/
	hipEvent_t start, stop;
	// hipEventCreate: 创建一个事件对象,异步启动
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: 记录一个事件,异步启动,start记录起始时间
	hipEventRecord(start, 0);

	size_t lengthA{ elements_num * sizeof(float) }, lengthB{ elements_num * sizeof(float) };
	float *d_A{ nullptr }, *d_B{ nullptr }, *d_partial_C{ nullptr };

	// hipMalloc: 在设备端分配内存
	hipMalloc(&d_A, lengthA);
	hipMalloc(&d_B, lengthB);

	/* hipMemcpy: 在主机端和设备端拷贝数据,此函数第四个参数仅能是下面之一:
	(1). hipMemcpyHostToHost: 拷贝数据从主机端到主机端
	(2). hipMemcpyHostToDevice: 拷贝数据从主机端到设备端
	(3). hipMemcpyDeviceToHost: 拷贝数据从设备端到主机端
	(4). hipMemcpyDeviceToDevice: 拷贝数据从设备端到设备端
	(5). hipMemcpyDefault: 从指针值自动推断拷贝数据方向,需要支持
	统一虚拟寻址(CUDA6.0及以上版本)
	hipMemcpy函数对于主机是同步的 */
	hipMemcpy(d_A, A, lengthA, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, lengthB, hipMemcpyHostToDevice);

	const int threadsPerBlock{ 256 };
	const int blocksPerGrid = std::min(64, (elements_num + threadsPerBlock - 1) / threadsPerBlock);
	size_t lengthC{ blocksPerGrid * sizeof(float) };
	hipMalloc(&d_partial_C, lengthC);

	/* <<< >>>: 为CUDA引入的运算符,指定线程网格和线程块维度等,传递执行参
	数给CUDA编译器和运行时系统,用于说明内核函数中的线程数量,以及线程是如何
	组织的;尖括号中这些参数并不是传递给设备代码的参数,而是告诉运行时如何
	启动设备代码,传递给设备代码本身的参数是放在圆括号中传递的,就像标准的函
	数调用一样;不同计算能力的设备对线程的总数和组织方式有不同的约束;必须
	先为kernel中用到的数组或变量分配好足够的空间,再调用kernel函数,否则在
	GPU计算时会发生错误,例如越界等;
	使用运行时API时,需要在调用的内核函数名与参数列表直接以<<<Dg,Db,Ns,S>>>
	的形式设置执行配置,其中：Dg是一个dim3型变量,用于设置grid的维度和各个
	维度上的尺寸.设置好Dg后,grid中将有Dg.x*Dg.y*Dg.z个block;Db是
	一个dim3型变量,用于设置block的维度和各个维度上的尺寸.设置好Db后,每个
	block中将有Db.x*Db.y*Db.z个thread;Ns是一个size_t型变量,指定各块为此调
	用动态分配的共享存储器大小,这些动态分配的存储器可供声明为外部数组
	(extern __shared__)的其他任何变量使用;Ns是一个可选参数,默认值为0;S为
	hipStream_t类型,用于设置与内核函数关联的流.S是一个可选参数,默认值0. */
	dot_product << < blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_partial_C, elements_num);

	/* hipDeviceSynchronize: kernel的启动是异步的, 为了定位它是否出错, 一
	般需要加上hipDeviceSynchronize函数进行同步; 将会一直处于阻塞状态,直到
	前面所有请求的任务已经被全部执行完毕,如果前面执行的某个任务失败,将会
	返回一个错误；当程序中有多个流,并且流之间在某一点需要通信时,那就必须
	在这一点处加上同步的语句,即hipDeviceSynchronize；异步启动
	reference: https://stackoverflow.com/questions/11888772/when-to-call-cudadevicesynchronize */
	//hipDeviceSynchronize();

	std::unique_ptr<float[]> partial_C(new float[blocksPerGrid]);
	hipMemcpy(partial_C.get(), d_partial_C, lengthC, hipMemcpyDeviceToHost);

	*value = 0.f;
	for (int i = 0; i < blocksPerGrid; ++i) {
		(*value) += partial_C[i];
	}

	// hipFree: 释放设备上由hipMalloc函数分配的内存
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_partial_C);

	// hipEventRecord: 记录一个事件,异步启动,stop记录结束时间
	hipEventRecord(stop, 0);
	// hipEventSynchronize: 事件同步,等待一个事件完成,异步启动
	hipEventSynchronize(stop);
	// cudaEventElapseTime: 计算两个事件之间经历的时间,单位为毫秒,异步启动
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: 销毁事件对象,异步启动
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}

static int dot_product_gpu_2(const float* A, const float* B, float* value, int elements_num, float* elapsed_time)
{
	// hipDeviceProp_t: cuda设备属性结构体
	hipDeviceProp_t prop;
	int count;
	// hipGetDeviceCount: 获得计算能力设备的数量
	hipGetDeviceCount(&count);
	//fprintf(stderr, "device count: %d\n", count);
	int whichDevice;
	// hipGetDevice: 获得当前正在使用的设备ID，设备ID从0开始编号
	hipGetDevice(&whichDevice);
	// hipGetDeviceProperties: 获取GPU设备相关信息
	hipGetDeviceProperties(&prop, whichDevice);
	// hipDeviceProp_t::canMapHostMemory: GPU是否支持设备映射主机内存
	if (prop.canMapHostMemory != 1) {
		fprintf(stderr, "Device cannot map memory.\n");
		return -1;
	}
	
	// hipSetDeviceFlags: 设置设备要用于执行的标志
	// 将设备置入能分配零拷贝内存的状态
	hipSetDeviceFlags(hipDeviceMapHost);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	const int threadsPerBlock{ 256 };
	const int blocksPerGrid = std::min(64, (elements_num + threadsPerBlock - 1) / threadsPerBlock);

	size_t lengthA{ elements_num * sizeof(float) }, lengthB{ elements_num * sizeof(float) };
	float *d_A{ nullptr }, *d_B{ nullptr }, *d_partial_C{ nullptr };
	float *a{ nullptr }, *b{ nullptr }, *partial_c{ nullptr };

	/* hipHostAlloc: 分配主机内存。C库函数malloc将分配标准的，可
	分页的(Pagable)主机内存，而hipHostAlloc将分配页锁定的主机内存。页锁定内
	存也称为固定内存(Pinned Memory)或者不可分页内存，它有一个重要的属性：操作系
	统将不会对这块内存分页并交换到磁盘上，从而确保了该内存始终驻留在物理内
	存中。因此，操作系统能够安全地使某个应用程序访问该内存的物理地址，因为
	这块内存将不会被破坏或者重新定位。由于GPU知道内存的物理地址，因此可以通
	过"直接内存访问(Direct Memory Access, DMA)"技术来在GPU和主机之间复制数据。
	固定内存是一把双刃剑。当使用固定内存时，你将失去虚拟内存的所有功能。
	建议：仅对hipMemcpy调用中的源内存或者目标内存，才使用页锁定内存，并且在
	不再需要使用它们时立即释放。
	零拷贝内存：通过hipHostAlloc函数+hipHostMallocMapped参数，而固定内存是
	hipHostAlloc函数+hipHostMallocDefault参数。通过hipHostMallocMapped分配
	的主机内存也是固定的，它与通过hipHostMallocDefault分配的固定内存有着相同
	的属性。但这种内存除了可以用于主机与GPU之间的内存复制外，还可以在CUDA C核
	函数中直接访问这种类型的主机内存，而不需要复制到GPU，因此也称为零拷贝内存。
	hipHostMallocMapped：这个标志告诉运行时将从GPU中访问这块内存。
	hipHostMallocWriteCombined：这个标志表示，运行时应该将内存分配为"合并式写
	入(Write-Combined)"内存。这个标志并不会改变应用程序的性能，但却可以显著地
	提升GPU读取内存时的性能。然而，当CPU也要读取这块内存时，"合并式写入"会显得
	很低效。
	对于集成GPU，使用零拷贝内存通常都会带来性能提升，因为内存在物理上与主机是
	共享的。将缓冲区声明为零拷贝内存的唯一作用就是避免不必要的数据复制。所有类型
	的固定内存都存在一定的局限性，零拷贝内存同样不例外：每个固定内存都会占用系统
	的可用物理内存，这最终将降低系统的性能。
	当输入内存和输出内存都只能使用一次时，那么在独立GPU上使用零拷贝内存将带来性能提升。 */
	// allocate the memory on the CPU
	hipHostAlloc(&a, lengthA, hipHostMallocWriteCombined | hipHostMallocMapped);
	hipHostAlloc(&b, lengthB, hipHostMallocWriteCombined | hipHostMallocMapped);
	hipHostAlloc(&partial_c, blocksPerGrid * sizeof(float), hipHostMallocMapped);

	/* hipHostGetDevicePointer: 获得由hipHostAlloc分配的映射主机内存的设备指针。
	由于GPU的虚拟内存空间地址映射与CPU不同，而hipHostAlloc返回的是CPU上的指针，
	因此需要调用hipHostGetDevicePointer函数来获得这块内存在GPU上的有效指针。这些指针
	将被传递给核函数，并在随后由GPU对这块内存执行读取和写入等操作 */
	// find out the GPU pointers
	hipHostGetDevicePointer(&d_A, a, 0);
	hipHostGetDevicePointer(&d_B, b, 0);
	hipHostGetDevicePointer(&d_partial_C, partial_c, 0);

	memcpy(a, A, lengthA);
	memcpy(b, B, lengthB);

	dot_product << < blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_partial_C, elements_num);

	/* hipDeviceSynchronize: 等待计算设备完成, 将CPU与GPU同步*/
	hipDeviceSynchronize();

	*value = 0.f;
	for (int i = 0; i < blocksPerGrid; ++i) {
		(*value) += partial_c[i];
	}

	// hipHostFree: 释放设备上由hipHostAlloc函数分配的内存
	hipHostFree(d_A);
	hipHostFree(d_B);
	hipHostFree(d_partial_C);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(elapsed_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}

int dot_product_gpu(const float* A, const float* B, float* value, int elements_num, float* elapsed_time)
{
	int ret{ 0 };
	//ret = dot_product_gpu_1(A, B, value, elements_num, elapsed_time); // 普通实现
	ret = dot_product_gpu_2(A, B, value, elements_num, elapsed_time); // 通过零拷贝内存实现

	return ret;
}
