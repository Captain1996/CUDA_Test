#include "hip/hip_runtime.h"
﻿#include "funset.hpp"
#include <iostream>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

// reference: C:\ProgramData\NVIDIA Corporation\CUDA Samples\v8.0\0_Simple\vectorAdd
/* __global__: 函数类型限定符;在设备上运行;在主机端调用,计算能力3.2及以上可以在
设备端调用;声明的函数的返回值必须是void类型;对此类型函数的调用是异步的,即在
设备完全完成它的运行之前就返回了;对此类型函数的调用必须指定执行配置,即用于在
设备上执行函数时的grid和block的维度,以及相关的流(即插入<<<   >>>运算符);
a kernel,表示此函数为内核函数(运行在GPU上的CUDA并行计算函数称为kernel(内核函
数),内核函数必须通过__global__函数类型限定符定义);*/
__global__ static void vector_add(const float *A, const float *B, float *C, int numElements)
{
	/* gridDim: 内置变量,用于描述线程网格的维度,对于所有线程块来说,这个
	变量是一个常数,用来保存线程格每一维的大小,即每个线程格中线程块的数量.
	一个grid为三维,为dim3类型；
	blockDim: 内置变量,用于说明每个block的维度与尺寸.为dim3类型,包含
	了block在三个维度上的尺寸信息;对于所有线程块来说,这个变量是一个常数,
	保存的是线程块中每一维的线程数量;
	blockIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程块的索引;用
	于说明当前thread所在的block在整个grid中的位置,blockIdx.x取值范围是
	[0,gridDim.x-1],blockIdx.y取值范围是[0, gridDim.y-1].为uint3类型,
	包含了一个block在grid中各个维度上的索引信息;
	threadIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程索引;用于
	说明当前thread在block中的位置;如果线程是一维的可获取threadIdx.x,如果
	是二维的还可获取threadIdx.y,如果是三维的还可获取threadIdx.z;为uint3类
	型,包含了一个thread在block中各个维度的索引信息 */
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numElements) {
		C[i] = A[i] + B[i];
	}
}

int vector_add_gpu(const float* A, const float* B, float* C, int numElements, float* elapsed_time)
{
	/* Error code to check return values for CUDA calls
	hipError_t: CUDA Error types, 枚举类型,CUDA错误码,成功返回
	hipSuccess(0),否则返回其它(>0) */
	hipError_t err{ hipSuccess };

	/* hipEvent_t: CUDA event types，结构体类型, CUDA事件，用于测量GPU在某
	个任务上花费的时间，CUDA中的事件本质上是一个GPU时间戳，由于CUDA事件是在
	GPU上实现的，因此它们不适于对同时包含设备代码和主机代码的混合代码计时*/
	hipEvent_t start, stop;
	// hipEventCreate: 创建一个事件对象，异步启动
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: 记录一个事件，异步启动,start记录起始时间
	hipEventRecord(start, 0);

	size_t length{ numElements * sizeof(float) };
	float *d_A{ nullptr }, *d_B{ nullptr }, *d_C{ nullptr };

	// hipMalloc: 在设备端分配内存
	err = hipMalloc(&d_A, length);
	if (err != hipSuccess) {
		// hipGetErrorString: 返回错误码的描述字符串
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
			hipGetErrorString(err));
		return -1;
	}
	err = hipMalloc(&d_B, length);
	if (err != hipSuccess) PRINT_ERROR_INFO(hipMalloc);
	err = hipMalloc(&d_C, length);
	if (err != hipSuccess) PRINT_ERROR_INFO(hipMalloc);

	/* hipMemcpy: 在主机端和设备端拷贝数据,此函数第四个参数仅能是下面之一:
	(1). hipMemcpyHostToHost: 拷贝数据从主机端到主机端
	(2). hipMemcpyHostToDevice: 拷贝数据从主机端到设备端
	(3). hipMemcpyDeviceToHost: 拷贝数据从设备端到主机端
	(4). hipMemcpyDeviceToDevice: 拷贝数据从设备端到设备端
	(5). hipMemcpyDefault: 从指针值自动推断拷贝数据方向,需要支持
	统一虚拟寻址(CUDA6.0及以上版本)
	hipMemcpy函数对于主机是同步的 */
	err = hipMemcpy(d_A, A, length, hipMemcpyHostToDevice);
	if (err != hipSuccess) PRINT_ERROR_INFO(hipMemcpy);
	err = hipMemcpy(d_B, B, length, hipMemcpyHostToDevice);
	if (err != hipSuccess) PRINT_ERROR_INFO(hipMemcpy);

	// Launch the Vector Add CUDA kernel
	const int threadsPerBlock{ 256 };
	const int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	fprintf(stderr, "CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	/* <<< >>>: 为CUDA引入的运算符,指定线程网格和线程块维度等,传递执行参
	数给CUDA编译器和运行时系统,用于说明内核函数中的线程数量,以及线程是如何
	组织的;尖括号中这些参数并不是传递给设备代码的参数,而是告诉运行时如何
	启动设备代码,传递给设备代码本身的参数是放在圆括号中传递的,就像标准的函
	数调用一样;不同计算能力的设备对线程的总数和组织方式有不同的约束;必须
	先为kernel中用到的数组或变量分配好足够的空间,再调用kernel函数,否则在
	GPU计算时会发生错误,例如越界等;
	使用运行时API时,需要在调用的内核函数名与参数列表直接以<<<Dg,Db,Ns,S>>>
	的形式设置执行配置,其中：Dg是一个dim3型变量,用于设置grid的维度和各个
	维度上的尺寸.设置好Dg后,grid中将有Dg.x*Dg.y*Dg.z个block;Db是
	一个dim3型变量,用于设置block的维度和各个维度上的尺寸.设置好Db后,每个
	block中将有Db.x*Db.y*Db.z个thread;Ns是一个size_t型变量,指定各块为此调
	用动态分配的共享存储器大小,这些动态分配的存储器可供声明为外部数组
	(extern __shared__)的其他任何变量使用;Ns是一个可选参数,默认值为0;S为
	hipStream_t类型,用于设置与内核函数关联的流.S是一个可选参数,默认值0. */
	vector_add << <blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_C, numElements);
	/* hipGetLastError: 在同一个主机线程中,返回运行时调用中产生的最后一个
	错误并将其重置为hipSuccess;此函数也可能返回以前异步启动的错误码;当有
	多个错误在对hipGetLastError的调用之间发生时,仅最后一个错误会被报告;
	kernel的启动是异步的,为了定位它是否出错,一般需要加上
	hipDeviceSynchronize函数进行同步,然后再调用hipGetLastError函数;*/
	err = hipGetLastError();
	if (err != hipSuccess) PRINT_ERROR_INFO(hipGetLastError);
	// Copy the device result vector in device memory to the host result vector in host memory.
	err = hipMemcpy(C, d_C, length, hipMemcpyDeviceToHost);
	if (err != hipSuccess) PRINT_ERROR_INFO(hipMemcpy);

	// hipFree: 释放设备上由hipMalloc函数分配的内存
	err = hipFree(d_A);
	if (err != hipSuccess) PRINT_ERROR_INFO(hipFree);
	err = hipFree(d_B);
	if (err != hipSuccess) PRINT_ERROR_INFO(hipFree);
	err = hipFree(d_C);
	if (err != hipSuccess) PRINT_ERROR_INFO(hipFree);

	// hipEventRecord: 记录一个事件，异步启动,stop记录结束时间
	hipEventRecord(stop, 0);
	// hipEventSynchronize: 事件同步，等待一个事件完成，异步启动
	hipEventSynchronize(stop);
	// cudaEventElapseTime: 计算两个事件之间经历的时间，单位为毫秒，异步启动
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: 销毁事件对象，异步启动
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return err;
}
