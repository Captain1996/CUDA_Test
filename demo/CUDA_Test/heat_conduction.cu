#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <algorithm>
#include <memory>
#include <vector>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

/* __global__: 函数类型限定符;在设备上运行;在主机端调用,计算能力3.2及以上可以在
设备端调用;声明的函数的返回值必须是void类型;对此类型函数的调用是异步的,即在
设备完全完成它的运行之前就返回了;对此类型函数的调用必须指定执行配置,即用于在
设备上执行函数时的grid和block的维度,以及相关的流(即插入<<<   >>>运算符);
a kernel,表示此函数为内核函数(运行在GPU上的CUDA并行计算函数称为kernel(内核函
数),内核函数必须通过__global__函数类型限定符定义); */
__global__ static void copy_const_kernel(float* iptr, const float* cptr)
{
	/* gridDim: 内置变量,用于描述线程网格的维度,对于所有线程块来说,这个
	变量是一个常数,用来保存线程格每一维的大小,即每个线程格中线程块的数量.
	一个grid为三维,为dim3类型；
	blockDim: 内置变量,用于说明每个block的维度与尺寸.为dim3类型,包含
	了block在三个维度上的尺寸信息;对于所有线程块来说,这个变量是一个常数,
	保存的是线程块中每一维的线程数量;
	blockIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程块的索引;用
	于说明当前thread所在的block在整个grid中的位置,blockIdx.x取值范围是
	[0,gridDim.x-1],blockIdx.y取值范围是[0, gridDim.y-1].为uint3类型,
	包含了一个block在grid中各个维度上的索引信息;
	threadIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程索引;用于
	说明当前thread在block中的位置;如果线程是一维的可获取threadIdx.x,如果
	是二维的还可获取threadIdx.y,如果是三维的还可获取threadIdx.z;为uint3类
	型,包含了一个thread在block中各个维度的索引信息 */
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	if (cptr[offset] != 0) iptr[offset] = cptr[offset];
}

__global__ static void blend_kernel(float* outSrc, const float* inSrc, int width, int height, float speed)
{
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	int left = offset - 1;
	int right = offset + 1;
	if (x == 0) ++left;
	if (x == width - 1) --right;

	int top = offset - height;
	int bottom = offset + height;
	if (y == 0) top += height;
	if (y == height - 1) bottom -= height;

	outSrc[offset] = inSrc[offset] + speed * (inSrc[top] + inSrc[bottom] + inSrc[left] + inSrc[right] - inSrc[offset] * 4);
}

/* __device__: 函数类型限定符,表明被修饰的函数在设备上执行，只能从设备上调用，
但只能在其它__device__函数或者__global__函数中调用；__device__函数不支持递归；
__device__函数的函数体内不能声明静态变量；__device__函数的参数数目是不可变化的;
不能对__device__函数取指针 */
__device__ static unsigned char value(float n1, float n2, int hue)
{
	if (hue > 360) hue -= 360;
	else if (hue < 0) hue += 360;

	if (hue < 60)
		return (unsigned char)(255 * (n1 + (n2 - n1)*hue / 60));
	if (hue < 180)
		return (unsigned char)(255 * n2);
	if (hue < 240)
		return (unsigned char)(255 * (n1 + (n2 - n1)*(240 - hue) / 60));
	return (unsigned char)(255 * n1);
}

__global__ static void float_to_color(unsigned char *optr, const float *outSrc)
{
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float l = outSrc[offset];
	float s = 1;
	int h = (180 + (int)(360.0f * outSrc[offset])) % 360;
	float m1, m2;

	if (l <= 0.5f) m2 = l * (1 + s);
	else m2 = l + s - l * s;
	m1 = 2 * l - m2;

	optr[offset * 4 + 0] = value(m1, m2, h + 120);
	optr[offset * 4 + 1] = value(m1, m2, h);
	optr[offset * 4 + 2] = value(m1, m2, h - 120);
	optr[offset * 4 + 3] = 255;
}

static int heat_conduction_gpu_1(unsigned char* ptr, int width, int height, const float* src, float speed, float* elapsed_time)
{
	/* hipEvent_t: CUDA event types,结构体类型, CUDA事件,用于测量GPU在某
	个任务上花费的时间,CUDA中的事件本质上是一个GPU时间戳,由于CUDA事件是在
	GPU上实现的,因此它们不适于对同时包含设备代码和主机代码的混合代码计时 */
	hipEvent_t start, stop;
	// hipEventCreate: 创建一个事件对象,异步启动
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: 记录一个事件,异步启动,start记录起始时间
	hipEventRecord(start, 0);

	float* dev_inSrc{ nullptr };
	float* dev_outSrc{ nullptr };
	float* dev_constSrc{ nullptr };
	unsigned char* dev_image{ nullptr };
	const size_t length1{ width * height * sizeof(float) };
	const size_t length2{ width * height * 4 * sizeof(unsigned char) };

	// hipMalloc: 在设备端分配内存
	hipMalloc(&dev_inSrc, length1);
	hipMalloc(&dev_outSrc, length1);
	hipMalloc(&dev_constSrc, length1);
	hipMalloc(&dev_image, length2);

	/* hipMemcpy: 在主机端和设备端拷贝数据,此函数第四个参数仅能是下面之一:
	(1). hipMemcpyHostToHost: 拷贝数据从主机端到主机端
	(2). hipMemcpyHostToDevice: 拷贝数据从主机端到设备端
	(3). hipMemcpyDeviceToHost: 拷贝数据从设备端到主机端
	(4). hipMemcpyDeviceToDevice: 拷贝数据从设备端到设备端
	(5). hipMemcpyDefault: 从指针值自动推断拷贝数据方向,需要支持
	统一虚拟寻址(CUDA6.0及以上版本)
	hipMemcpy函数对于主机是同步的 */
	hipMemcpy(dev_constSrc, src, length1, hipMemcpyHostToDevice);

	const int threads_block{ 16 };
	/* dim3: 基于uint3定义的内置矢量类型，相当于由3个unsigned int类型组成的
	结构体，可表示一个三维数组，在定义dim3类型变量时，凡是没有赋值的元素都
	会被赋予默认值1 */
	dim3 blocks(width / threads_block, height / threads_block);
	dim3 threads(threads_block, threads_block);

	for (int i = 0; i < 90; ++i) {
		copy_const_kernel << <blocks, threads >> >(dev_inSrc, dev_constSrc);
		blend_kernel << <blocks, threads >> >(dev_outSrc, dev_inSrc, width, height, speed);
		std::swap(dev_inSrc, dev_outSrc);
	}

	/* <<< >>>: 为CUDA引入的运算符,指定线程网格和线程块维度等,传递执行参
	数给CUDA编译器和运行时系统,用于说明内核函数中的线程数量,以及线程是如何
	组织的;尖括号中这些参数并不是传递给设备代码的参数,而是告诉运行时如何
	启动设备代码,传递给设备代码本身的参数是放在圆括号中传递的,就像标准的函
	数调用一样;不同计算能力的设备对线程的总数和组织方式有不同的约束;必须
	先为kernel中用到的数组或变量分配好足够的空间,再调用kernel函数,否则在
	GPU计算时会发生错误,例如越界等;
	使用运行时API时,需要在调用的内核函数名与参数列表直接以<<<Dg,Db,Ns,S>>>
	的形式设置执行配置,其中：Dg是一个dim3型变量,用于设置grid的维度和各个
	维度上的尺寸.设置好Dg后,grid中将有Dg.x*Dg.y*Dg.z个block;Db是
	一个dim3型变量,用于设置block的维度和各个维度上的尺寸.设置好Db后,每个
	block中将有Db.x*Db.y*Db.z个thread;Ns是一个size_t型变量,指定各块为此调
	用动态分配的共享存储器大小,这些动态分配的存储器可供声明为外部数组
	(extern __shared__)的其他任何变量使用;Ns是一个可选参数,默认值为0;S为
	hipStream_t类型,用于设置与内核函数关联的流.S是一个可选参数,默认值0. */
	float_to_color << <blocks, threads >> >(dev_image, dev_inSrc);

	hipMemcpy(ptr, dev_image, length2, hipMemcpyDeviceToHost);

	// hipFree: 释放设备上由hipMalloc函数分配的内存
	hipFree(dev_inSrc);
	hipFree(dev_outSrc);
	hipFree(dev_constSrc);
	hipFree(dev_image);

	// hipEventRecord: 记录一个事件,异步启动,stop记录结束时间
	hipEventRecord(stop, 0);
	// hipEventSynchronize: 事件同步,等待一个事件完成,异步启动
	hipEventSynchronize(stop);
	// cudaEventElapseTime: 计算两个事件之间经历的时间,单位为毫秒,异步启动
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: 销毁事件对象,异步启动
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}

static int heat_conduction_gpu_2(unsigned char* ptr, int width, int height, const float* src, float speed, float* elapsed_time)
{
	return 0;
}

static int heat_conduction_gpu_3(unsigned char* ptr, int width, int height, const float* src, float speed, float* elapsed_time)
{
	return 0;
}

int heat_conduction_gpu(unsigned char* ptr, int width, int height, const float* src, float speed, float* elapsed_time)
{
	int ret{ 0 };
	ret = heat_conduction_gpu_1(ptr, width, height, src, speed, elapsed_time); // 没有采用纹理内存
	//ret = heat_conduction_gpu_2(ptr, width, height, src, speed, elapsed_time); // 采用一维纹理内存
	//ret = heat_conduction_gpu_3(ptr, width, height, src, speed, elapsed_time); // 采用二维纹理内存

	return ret;
}
