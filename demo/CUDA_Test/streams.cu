#include "hip/hip_runtime.h"
﻿#include "funset.hpp"
#include <iostream>
#include <algorithm>
#include <memory>
#include <vector>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

/* __global__: 函数类型限定符;在设备上运行;在主机端调用,计算能力3.2及以上可以在
设备端调用;声明的函数的返回值必须是void类型;对此类型函数的调用是异步的,即在
设备完全完成它的运行之前就返回了;对此类型函数的调用必须指定执行配置,即用于在
设备上执行函数时的grid和block的维度,以及相关的流(即插入<<<   >>>运算符);
a kernel,表示此函数为内核函数(运行在GPU上的CUDA并行计算函数称为kernel(内核函
数),内核函数必须通过__global__函数类型限定符定义); */
__global__ static void stream_kernel(int* a, int* b, int* c, int length)
{
	/* gridDim: 内置变量,用于描述线程网格的维度,对于所有线程块来说,这个
	变量是一个常数,用来保存线程格每一维的大小,即每个线程格中线程块的数量.
	一个grid为三维,为dim3类型；
	blockDim: 内置变量,用于说明每个block的维度与尺寸.为dim3类型,包含
	了block在三个维度上的尺寸信息;对于所有线程块来说,这个变量是一个常数,
	保存的是线程块中每一维的线程数量;
	blockIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程块的索引;用
	于说明当前thread所在的block在整个grid中的位置,blockIdx.x取值范围是
	[0,gridDim.x-1],blockIdx.y取值范围是[0, gridDim.y-1].为uint3类型,
	包含了一个block在grid中各个维度上的索引信息;
	threadIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程索引;用于
	说明当前thread在block中的位置;如果线程是一维的可获取threadIdx.x,如果
	是二维的还可获取threadIdx.y,如果是三维的还可获取threadIdx.z;为uint3类
	型,包含了一个thread在block中各个维度的索引信息 */
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < length) {
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}

int streams_gpu_1(const int* a, const int* b, int* c, int length, float* elapsed_time)
{
	// hipDeviceProp_t: cuda设备属性结构体
	hipDeviceProp_t prop;
	// hipGetDeviceProperties: 获取GPU设备相关信息
	hipGetDeviceProperties(&prop, 0);
	/* hipDeviceProp_t::deviceOverlap: GPU是否支持设备重叠(Device Overlap)功
	能,支持设备重叠功能的GPU能够在执行一个CUDA C核函数的同时，还能在设备与
	主机之间执行复制等操作 */
	if (!prop.deviceOverlap) {
		printf("Device will not handle overlaps, so no speed up from streams\n");
		return -1;
	}

	/* hipEvent_t: CUDA event types,结构体类型, CUDA事件,用于测量GPU在某
	个任务上花费的时间,CUDA中的事件本质上是一个GPU时间戳,由于CUDA事件是在
	GPU上实现的,因此它们不适于对同时包含设备代码和主机代码的混合代码计时 */
	hipEvent_t start, stop;
	// hipEventCreate: 创建一个事件对象,异步启动
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: 记录一个事件,异步启动,start记录起始时间
	hipEventRecord(start, 0);

	/* hipStream_t: cuda 流，结构体类型, CUDA流表示一个GPU操作队列，并且该
	队列中的操作将以指定的顺序执行。可以将每个流视为GPU上的一个任务，并且这
	些任务可以并行执行。 */
	hipStream_t stream;
	// hipStreamCreate: 初始化流，创建一个新的异步流
	hipStreamCreate(&stream);

	int *host_a{ nullptr }, *host_b{ nullptr }, *host_c{ nullptr };
	int *dev_a{ nullptr }, *dev_b{ nullptr }, *dev_c{ nullptr };
	const int N{ length / 20 };

	// hipMalloc: 在设备端分配内存
	hipMalloc(&dev_a, N * sizeof(int));
	hipMalloc(&dev_b, N * sizeof(int));
	hipMalloc(&dev_c, N * sizeof(int));
	/* hipHostAlloc: 分配主机内存(固定内存)。C库函数malloc将分配标准的，可
	分页的(Pagable)主机内存，而hipHostAlloc将分配页锁定的主机内存。页锁定内
	存也称为固定内存(Pinned Memory)或者不可分页内存，它有一个重要的属性：操作系
	统将不会对这块内存分页并交换到磁盘上，从而确保了该内存始终驻留在物理内
	存中。因此，操作系统能够安全地使某个应用程序访问该内存的物理地址，因为
	这块内存将不会被破坏或者重新定位。由于GPU知道内存的物理地址，因此可以通
	过"直接内存访问(Direct Memory Access, DMA)"技术来在GPU和主机之间复制数据。
	固定内存是一把双刃剑。当使用固定内存时，你将失去虚拟内存的所有功能。
	建议：仅对hipMemcpy调用中的源内存或者目标内存，才使用页锁定内存，并且在
	不再需要使用它们时立即释放。 */
	// 分配由流使用的页锁定内存
	hipHostAlloc(&host_a, length * sizeof(int), hipHostMallocDefault);
	hipHostAlloc(&host_b, length * sizeof(int), hipHostMallocDefault);
	hipHostAlloc(&host_c, length * sizeof(int), hipHostMallocDefault);

	//for (int i = 0; i < length; ++i) {
	//	host_a[i] = a[i];
	//	host_b[i] = b[i];
	//}
	memcpy(host_a, a, length * sizeof(int));
	memcpy(host_b, b, length * sizeof(int));

	for (int i = 0; i < length; i += N) {
		/* hipMemcpyAsync: 在GPU与主机之间复制数据。hipMemcpy的行为类
		似于C库函数memcpy。尤其是，这个函数将以同步方式执行，这意味着，
		当函数返回时，复制操作就已经完成，并且在输出缓冲区中包含了复制
		进去的内容。异步函数的行为与同步函数相反，在调用hipMemcpyAsync时，
		只是放置了一个请求，表示在流中执行一次内存复制操作，这个流是通过
		参数stream来指定的。当函数返回时，我们无法确保复制操作是否已经
		启动，更无法保证它们是否已经结束。我们能够得到的保证是，复制操作肯定
		会当下一个被放入流中的操作之前执行。任何传递给hipMemcpyAsync的主机
		内存指针都必须已经通过hipHostAlloc分配好内存。也就是，你只能以异步
		方式对页锁定内存进行复制操作 */
		// 将锁定内存以异步方式复制到设备上
		hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);

		/* <<< >>>: 为CUDA引入的运算符,指定线程网格和线程块维度等,传递执行参
		数给CUDA编译器和运行时系统,用于说明内核函数中的线程数量,以及线程是如何
		组织的;尖括号中这些参数并不是传递给设备代码的参数,而是告诉运行时如何
		启动设备代码,传递给设备代码本身的参数是放在圆括号中传递的,就像标准的函
		数调用一样;不同计算能力的设备对线程的总数和组织方式有不同的约束;必须
		先为kernel中用到的数组或变量分配好足够的空间,再调用kernel函数,否则在
		GPU计算时会发生错误,例如越界等;
		使用运行时API时,需要在调用的内核函数名与参数列表直接以<<<Dg,Db,Ns,S>>>
		的形式设置执行配置,其中：Dg是一个dim3型变量,用于设置grid的维度和各个
		维度上的尺寸.设置好Dg后,grid中将有Dg.x*Dg.y*Dg.z个block;Db是
		一个dim3型变量,用于设置block的维度和各个维度上的尺寸.设置好Db后,每个
		block中将有Db.x*Db.y*Db.z个thread;Ns是一个size_t型变量,指定各块为此调
		用动态分配的共享存储器大小,这些动态分配的存储器可供声明为外部数组
		(extern __shared__)的其他任何变量使用;Ns是一个可选参数,默认值为0;S为
		hipStream_t类型,用于设置与内核函数关联的流.S是一个可选参数,默认值0. */
		stream_kernel << <N / 256, 256, 0, stream >> >(dev_a, dev_b, dev_c, N);

		hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
	}

	/* hipStreamSynchronize: 等待传入流中的操作完成，主机在继续执行之前，要
	等待GPU执行完成 */
	hipStreamSynchronize(stream);

	//for (int i = 0; i < length; ++i)
	//	c[i] = host_c[i];
	memcpy(c, host_c, length * sizeof(int));

	// hipHostFree: 释放设备上由hipHostAlloc函数分配的内存
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	// hipFree: 释放设备上由hipMalloc函数分配的内存
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	// hipStreamDestroy: 销毁流
	hipStreamDestroy(stream);

	// hipEventRecord: 记录一个事件,异步启动,stop记录结束时间
	hipEventRecord(stop, 0);
	// hipEventSynchronize: 事件同步,等待一个事件完成,异步启动
	hipEventSynchronize(stop);
	// cudaEventElapseTime: 计算两个事件之间经历的时间,单位为毫秒,异步启动
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: 销毁事件对象,异步启动
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}

int streams_gpu_2(const int* a, const int* b, int* c, int length, float* elapsed_time)
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	if (!prop.deviceOverlap) {
		printf("Device will not handle overlaps, so no speed up from streams\n");
		return -1;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	int *host_a{ nullptr }, *host_b{ nullptr }, *host_c{ nullptr };
	int *dev_a0{ nullptr }, *dev_b0{ nullptr }, *dev_c0{ nullptr };
	int *dev_a1{ nullptr }, *dev_b1{ nullptr }, *dev_c1{ nullptr };
	const int N{ length / 20 };

	hipMalloc(&dev_a0, N * sizeof(int));
	hipMalloc(&dev_b0, N * sizeof(int));
	hipMalloc(&dev_c0, N * sizeof(int));
	hipMalloc(&dev_a1, N * sizeof(int));
	hipMalloc(&dev_b1, N * sizeof(int));
	hipMalloc(&dev_c1, N * sizeof(int));
	hipHostAlloc(&host_a, length * sizeof(int), hipHostMallocDefault);
	hipHostAlloc(&host_b, length * sizeof(int), hipHostMallocDefault);
	hipHostAlloc(&host_c, length * sizeof(int), hipHostMallocDefault);

	memcpy(host_a, a, length * sizeof(int));
	memcpy(host_b, b, length * sizeof(int));

	for (int i = 0; i < length; i += N * 2) {
		//hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		//hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		//stream_kernel << <N / 256, 256, 0, stream0 >> >(dev_a0, dev_b0, dev_c0, N);
		//hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);

		//hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		//hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		//stream_kernel << <N / 256, 256, 0, stream1 >> >(dev_a1, dev_b1, dev_c1, N);
		//hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);

		// 推荐采用宽度优先方式
		hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);

		hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);

		stream_kernel << <N / 256, 256, 0, stream0 >> >(dev_a0, dev_b0, dev_c0, N);
		stream_kernel << <N / 256, 256, 0, stream1 >> >(dev_a1, dev_b1, dev_c1, N);

		hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
	}

	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	memcpy(c, host_c, length * sizeof(int));

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a0);
	hipFree(dev_b0);
	hipFree(dev_c0);
	hipFree(dev_a1);
	hipFree(dev_b1);
	hipFree(dev_c1);
	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(elapsed_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}

int streams_gpu(const int* a, const int* b, int* c, int length, float* elapsed_time)
{
	int ret{ 0 };
	//ret = streams_gpu_1(a, b, c, length, elapsed_time); // 使用单个流
	ret = streams_gpu_2(a, b, c, length, elapsed_time); // 使用多个流

	return ret;
}
