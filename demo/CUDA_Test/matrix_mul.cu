#include "hip/hip_runtime.h"
﻿#include "funset.hpp"
#include <iostream>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

// reference: C:\ProgramData\NVIDIA Corporation\CUDA Samples\v8.0\0_Simple\matrixMul
/* __global__: 函数类型限定符;在设备上运行;在主机端调用,计算能力3.2及以上可以在
设备端调用;声明的函数的返回值必须是void类型;对此类型函数的调用是异步的,即在
设备完全完成它的运行之前就返回了;对此类型函数的调用必须指定执行配置,即用于在
设备上执行函数时的grid和block的维度,以及相关的流(即插入<<<   >>>运算符);
a kernel,表示此函数为内核函数(运行在GPU上的CUDA并行计算函数称为kernel(内核函
数),内核函数必须通过__global__函数类型限定符定义);*/
template <int BLOCK_SIZE>
__global__ static void matrix_mul(const float* A, const float* B, float* C, int wA, int wB)
{
	/* gridDim: 内置变量,用于描述线程网格的维度,对于所有线程块来说,这个
	变量是一个常数,用来保存线程格每一维的大小,即每个线程格中线程块的数量.
	一个grid为三维,为dim3类型；
	blockDim: 内置变量,用于说明每个block的维度与尺寸.为dim3类型,包含
	了block在三个维度上的尺寸信息;对于所有线程块来说,这个变量是一个常数,
	保存的是线程块中每一维的线程数量;
	blockIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程块的索引;用
	于说明当前thread所在的block在整个grid中的位置,blockIdx.x取值范围是
	[0,gridDim.x-1],blockIdx.y取值范围是[0, gridDim.y-1].为uint3类型,
	包含了一个block在grid中各个维度上的索引信息;
	threadIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程索引;用于
	说明当前thread在block中的位置;如果线程是一维的可获取threadIdx.x,如果
	是二维的还可获取threadIdx.y,如果是三维的还可获取threadIdx.z;为uint3类
	型,包含了一个thread在block中各个维度的索引信息 */
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;
	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index of the first sub-matrix of A processed by the block
	int aBegin = wA * BLOCK_SIZE * by;
	// Index of the last sub-matrix of A processed by the block
	int aEnd = aBegin + wA - 1;
	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;
	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;
	// Step size used to iterate through the sub-matrices of B
	int bStep = BLOCK_SIZE * wB;
	// Csub is used to store the element of the block sub-matrix that is computed by the thread
	float Csub = 0;

	// Loop over all the sub-matrices of A and B required to compute the block sub-matrix
	for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
		/* __shared__: 变量类型限定符；使用__shared__限定符，或者与__device__限
		定符连用，此时声明的变量位于block中的共享存储器空间中，与block具有相同
		的生命周期，仅可通过block内的所有线程访问；__shared__和__constant__变量
		默认为是静态存储；在__shared__前可以加extern关键字，但表示的是变量大小
		由执行参数确定；__shared__变量在声明时不能初始化；可以将CUDA C的关键字
		__shared__添加到变量声明中，这将使这个变量驻留在共享内存中；CUDA C编译
		器对共享内存中的变量与普通变量将分别采取不同的处理方式 */
		// Declaration of the shared memory array As used to store the sub-matrix of A
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		// Declaration of the shared memory array Bs used to store the sub-matrix of B
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from device memory to shared memory; each thread loads one element of each matrix
		As[ty][tx] = A[a + wA * ty + tx];
		Bs[ty][tx] = B[b + wB * ty + tx];

		/* __syncthreads: 对线程块中的线程进行同步；CUDA架构将确保，除非线程块
		中的每个线程都执行了__syncthreads()，否则没有任何线程能执行
		__syncthreads()之后的指令;在同一个block中的线程通过共享存储器(shared
		memory)交换数据，并通过栅栏同步(可以在kernel函数中需要同步的位置调用
		__syncthreads()函数)保证线程间能够正确地共享数据；使用clock()函数计时，
		在内核函数中要测量的一段代码的开始和结束的位置分别调用一次clock()函数，
		并将结果记录下来。由于调用__syncthreads()函数后，一个block中的所有
		thread需要的时间是相同的，因此只需要记录每个block执行需要的时间就行了，
		而不需要记录每个thread的时间 */
		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		/* reference:
			https://devblogs.nvidia.com/parallelforall/new-compiler-features-cuda-8/
			https://stackoverflow.com/questions/22278631/what-does-pragma-unroll-do-exactly-does-it-affect-the-number-of-threads/22279341
		编译器默认情况下将循环展开小的次数，#pragma unroll能够指定循环
		以多少次展开(程序员必须保证按这个展开是正确的)，pragma unroll 后
		必须紧接着处理的循环，可选择在其后接一个数字，指定必须展开多少次循环，
		#pragma unroll 1 表示禁止编译器将循环展开。如果没指定次数，对于常数
		次的循环，循环将完全展开，对于不确定次数的循环，循环将不会展开。
		*/
#pragma unroll
		// Multiply the two matrices together; each thread computes one element of the block sub-matrix
		for (int k = 0; k < BLOCK_SIZE; ++k) {
			Csub += As[ty][k] * Bs[k][tx];
		}

		// Synchronize to make sure that the preceding computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to device memory; each thread writes one element
	int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	C[c + wB * ty + tx] = Csub;
}

__global__ static void matrix_mul(const float* A, const float* B, float* C, int colsA, int rowsA, int colsB, int rowsB)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float sum{ 0.f };
	for (int t = 0; t < colsA; ++t) {
		sum += A[y * colsA + t] * B[t * colsB + x];
	}

	C[offset] = sum;
}

int matrix_mul_gpu(const float* A, const float* B, float* C, int colsA, int rowsA, int colsB, int rowsB, float* elapsed_time)
{
	CHECK(colsA == rowsB);

	/* hipEvent_t: CUDA event types，结构体类型, CUDA事件，用于测量GPU在某
	个任务上花费的时间，CUDA中的事件本质上是一个GPU时间戳，由于CUDA事件是在
	GPU上实现的，因此它们不适于对同时包含设备代码和主机代码的混合代码计时*/
	hipEvent_t start, stop;
	// hipEventCreate: 创建一个事件对象，异步启动
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: 记录一个事件，异步启动,start记录起始时间
	hipEventRecord(start, 0);

	size_t lengthA{ colsA * rowsA * sizeof(float) }, lengthB{ colsB * rowsB * sizeof(float) };
	size_t lengthC{ rowsA * colsB * sizeof(float) };
	float *d_A{ nullptr }, *d_B{ nullptr }, *d_C{ nullptr };

	// hipMalloc: 在设备端分配内存
	hipMalloc(&d_A, lengthA);
	hipMalloc(&d_B, lengthB);
	hipMalloc(&d_C, lengthC);

	/* hipMemcpy: 在主机端和设备端拷贝数据,此函数第四个参数仅能是下面之一:
	(1). hipMemcpyHostToHost: 拷贝数据从主机端到主机端
	(2). hipMemcpyHostToDevice: 拷贝数据从主机端到设备端
	(3). hipMemcpyDeviceToHost: 拷贝数据从设备端到主机端
	(4). hipMemcpyDeviceToDevice: 拷贝数据从设备端到设备端
	(5). hipMemcpyDefault: 从指针值自动推断拷贝数据方向,需要支持
	统一虚拟寻址(CUDA6.0及以上版本)
	hipMemcpy函数对于主机是同步的 */
	hipMemcpy(d_A, A, lengthA, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, lengthB, hipMemcpyHostToDevice);
	//hipMemcpy(d_C, C, lengthC, hipMemcpyHostToDevice);

	const int block_size{ 32 };
	/* dim3: 基于uint3定义的内置矢量类型，相当于由3个unsigned int类型组成的
	结构体，可表示一个三维数组，在定义dim3类型变量时，凡是没有赋值的元素都
	会被赋予默认值1 */
	dim3 dimsA(colsA, rowsA, 1);
	dim3 dimsB(colsB, rowsB, 1);
	CHECK(dimsA.x == dimsB.y);
	//fprintf(stderr, "MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);

	dim3 threads(block_size, block_size);
	dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

	/* <<< >>>: 为CUDA引入的运算符,指定线程网格和线程块维度等,传递执行参
	数给CUDA编译器和运行时系统,用于说明内核函数中的线程数量,以及线程是如何
	组织的;尖括号中这些参数并不是传递给设备代码的参数,而是告诉运行时如何
	启动设备代码,传递给设备代码本身的参数是放在圆括号中传递的,就像标准的函
	数调用一样;不同计算能力的设备对线程的总数和组织方式有不同的约束;必须
	先为kernel中用到的数组或变量分配好足够的空间,再调用kernel函数,否则在
	GPU计算时会发生错误,例如越界等;
	使用运行时API时,需要在调用的内核函数名与参数列表直接以<<<Dg,Db,Ns,S>>>
	的形式设置执行配置,其中：Dg是一个dim3型变量,用于设置grid的维度和各个
	维度上的尺寸.设置好Dg后,grid中将有Dg.x*Dg.y*Dg.z个block;Db是
	一个dim3型变量,用于设置block的维度和各个维度上的尺寸.设置好Db后,每个
	block中将有Db.x*Db.y*Db.z个thread;Ns是一个size_t型变量,指定各块为此调
	用动态分配的共享存储器大小,这些动态分配的存储器可供声明为外部数组
	(extern __shared__)的其他任何变量使用;Ns是一个可选参数,默认值为0;S为
	hipStream_t类型,用于设置与内核函数关联的流.S是一个可选参数,默认值0. */
	matrix_mul<block_size> <<< grid, threads >>>(d_A, d_B, d_C, dimsA.x, dimsB.x); // 运行较快
	//matrix_mul<< < grid, threads >> >(d_A, d_B, d_C, colsA, rowsA, colsB, rowsB);

	/* hipDeviceSynchronize: kernel的启动是异步的, 为了定位它是否出错, 一
	般需要加上hipDeviceSynchronize函数进行同步; 将会一直处于阻塞状态，直到
	前面所有请求的任务已经被全部执行完毕，如果前面执行的某个任务失败，将会
	返回一个错误；当程序中有多个流，并且流之间在某一点需要通信时，那就必须
	在这一点处加上同步的语句，即hipDeviceSynchronize；异步启动
	reference: https://stackoverflow.com/questions/11888772/when-to-call-cudadevicesynchronize */
	//hipDeviceSynchronize();

	hipMemcpy(C, d_C, lengthC, hipMemcpyDeviceToHost);
	// hipFree: 释放设备上由hipMalloc函数分配的内存
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// hipEventRecord: 记录一个事件，异步启动,stop记录结束时间
	hipEventRecord(stop, 0);
	// hipEventSynchronize: 事件同步，等待一个事件完成，异步启动
	hipEventSynchronize(stop);
	// cudaEventElapseTime: 计算两个事件之间经历的时间，单位为毫秒，异步启动
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: 销毁事件对象，异步启动
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}

