#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>
#include "common.hpp"

/* __global__: 函数类型限定符;在设备上运行;在主机端调用,计算能力3.2及以上可以在
设备端调用;声明的函数的返回值必须是void类型;对此类型函数的调用是异步的,即在
设备完全完成它的运行之前就返回了;对此类型函数的调用必须指定执行配置,即用于在
设备上执行函数时的grid和block的维度,以及相关的流(即插入<<<   >>>运算符);
a kernel,表示此函数为内核函数(运行在GPU上的CUDA并行计算函数称为kernel(内核函
数),内核函数必须通过__global__函数类型限定符定义);*/
__global__ static void bgr2bgr565(const unsigned char* src, int width, int height, unsigned char* dst)
{
	/* gridDim: 内置变量,用于描述线程网格的维度,对于所有线程块来说,这个
	变量是一个常数,用来保存线程格每一维的大小,即每个线程格中线程块的数量.
	一个grid为三维,为dim3类型；
	blockDim: 内置变量,用于说明每个block的维度与尺寸.为dim3类型,包含
	了block在三个维度上的尺寸信息;对于所有线程块来说,这个变量是一个常数,
	保存的是线程块中每一维的线程数量;
	blockIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程块的索引;用
	于说明当前thread所在的block在整个grid中的位置,blockIdx.x取值范围是
	[0,gridDim.x-1],blockIdx.y取值范围是[0, gridDim.y-1].为uint3类型,
	包含了一个block在grid中各个维度上的索引信息;
	threadIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程索引;用于
	说明当前thread在block中的位置;如果线程是一维的可获取threadIdx.x,如果
	是二维的还可获取threadIdx.y,如果是三维的还可获取threadIdx.z;为uint3类 
	型,包含了一个thread在block中各个维度的索引信息 */
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	//if (x == 0 && y == 0) {
	//	printf("%d, %d\n", width, height);
	//}

	if (x < width && y < height) { 
		const unsigned char* p = src + (y * width * 3 + x * 3);
		((unsigned short*)dst)[y * width + x] = (unsigned short)((p[0] >> 3) | ((p[1] & ~3) << 3) | ((p[2] & ~7) << 8));
	}
}

int bgr2bgr565_gpu(const unsigned char* src, int width, int height, unsigned char* dst, float* elapsed_time)
{
	unsigned char *dev_src{ nullptr }, *dev_dst{ nullptr };
	// hipMalloc: 在设备端分配内存
	hipMalloc(&dev_src, width * height * 3 * sizeof(unsigned char));
	hipMalloc(&dev_dst, width * height * 2 * sizeof(unsigned char));
	/* hipMemcpy: 在主机端和设备端拷贝数据,此函数第四个参数仅能是下面之一:
	(1). hipMemcpyHostToHost: 拷贝数据从主机端到主机端
	(2). hipMemcpyHostToDevice: 拷贝数据从主机端到设备端
	(3). hipMemcpyDeviceToHost: 拷贝数据从设备端到主机端
	(4). hipMemcpyDeviceToDevice: 拷贝数据从设备端到设备端
	(5). hipMemcpyDefault: 从指针值自动推断拷贝数据方向,需要支持
	统一虚拟寻址(CUDA6.0及以上版本)
	hipMemcpy函数对于主机是同步的 */
	hipMemcpy(dev_src, src, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
	/* hipMemset: 存储器初始化函数,在GPU内存上执行。用指定的值初始化或设置
	设备内存 */
	hipMemset(dev_dst, 0, width * height * 2 * sizeof(unsigned char));

	TIME_START_GPU

	/* dim3: 基于uint3定义的内置矢量类型，相当于由3个unsigned int类型组成的
	结构体，可表示一个三维数组，在定义dim3类型变量时，凡是没有赋值的元素都
	会被赋予默认值1 */
	// Note：每一个线程块支持的最大线程数量为1024，即threads.x*threads.y必须小于等于1024
	dim3 threads(32, 32);
	dim3 blocks((width + 31) / 32, (height + 31) / 32);

	/* <<< >>>: 为CUDA引入的运算符,指定线程网格和线程块维度等,传递执行参
	数给CUDA编译器和运行时系统,用于说明内核函数中的线程数量,以及线程是如何
	组织的;尖括号中这些参数并不是传递给设备代码的参数,而是告诉运行时如何
	启动设备代码,传递给设备代码本身的参数是放在圆括号中传递的,就像标准的函
	数调用一样;不同计算能力的设备对线程的总数和组织方式有不同的约束;必须
	先为kernel中用到的数组或变量分配好足够的空间,再调用kernel函数,否则在
	GPU计算时会发生错误,例如越界等 ;
	使用运行时API时,需要在调用的内核函数名与参数列表直接以<<<Dg,Db,Ns,S>>>
	的形式设置执行配置,其中：Dg是一个dim3型变量,用于设置grid的维度和各个
	维度上的尺寸.设置好Dg后,grid中将有Dg.x*Dg.y*Dg.z个block;Db是
	一个dim3型变量,用于设置block的维度和各个维度上的尺寸.设置好Db后,每个
	block中将有Db.x*Db.y*Db.z个thread;Ns是一个size_t型变量,指定各块为此调
	用动态分配的共享存储器大小,这些动态分配的存储器可供声明为外部数组
	(extern __shared__)的其他任何变量使用;Ns是一个可选参数,默认值为0;S为
	hipStream_t类型,用于设置与内核函数关联的流.S是一个可选参数,默认值0. */
	// Note: 核函数不支持传入参数为vector的data()指针，需要hipMalloc和hipMemcpy，因为vector是在主机内存中
	bgr2bgr565 << <blocks, threads >> >(dev_src, width, height, dev_dst);

	/* hipDeviceSynchronize: kernel的启动是异步的, 为了定位它是否出错, 一
	般需要加上hipDeviceSynchronize函数进行同步; 将会一直处于阻塞状态,直到
	前面所有请求的任务已经被全部执行完毕,如果前面执行的某个任务失败,将会
	返回一个错误；当程序中有多个流,并且流之间在某一点需要通信时,那就必须
	在这一点处加上同步的语句,即hipDeviceSynchronize；异步启动
	reference: https://stackoverflow.com/questions/11888772/when-to-call-cudadevicesynchronize */
	hipDeviceSynchronize();

	TIME_END_GPU

	hipMemcpy(dst, dev_dst, width * height * 2 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	// hipFree: 释放设备上由hipMalloc函数分配的内存
	hipFree(dev_dst);
	hipFree(dev_src);

	return 0;
}
