#include "hip/hip_runtime.h"
﻿#include "funset.hpp"
#include <iostream>
#include <algorithm>
#include <memory>
#include <vector>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

// 通过一个数据结构对球面建模
struct Sphere {
	float r, b, g;
	float radius;
	float x, y, z;
	/* __device__: 函数类型限定符,表明被修饰的函数在设备上执行，只能从设备上调用，
	但只能在其它__device__函数或者__global__函数中调用；__device__函数不支持递归；
	__device__函数的函数体内不能声明静态变量；__device__函数的参数数目是不可变化的;
	不能对__device__函数取指针 */
	__device__ float hit(float ox, float oy, float *n)
	{
		float dx = ox - x;
		float dy = oy - y;
		if (dx*dx + dy*dy < radius*radius) {
			float dz = sqrtf(radius*radius - dx*dx - dy*dy);
			*n = dz / sqrtf(radius * radius);
			return dz + z;
		}
		return -INF;
	}
};

// method2: 使用常量内存
/* __constant__: 变量类型限定符，或者与__device__限定符连用，这样声明的变量：存
在于常数存储器空间；与应用程序具有相同的生命周期；可以通过运行时库从主机端访问，
设备端的所有线程也可访问。__constant__变量默认为是静态存储。__constant__不能用
extern关键字声明为外部变量。__constant__变量只能在文件作用域中声明，不能再函数
体内声明。__constant__变量不能从device中赋值，只能从host中通过host运行时函数赋
值。__constant__将把变量的访问限制为只读。与从全局内存中读取数据相比，从常量内
存中读取相同的数据可以节约内存带宽。常量内存用于保存在核函数执行期间不会发生变
化的数据。
常量内存：用于保存在核函数执行期间不会发生变化的数据。NVIDIA硬件提供了64KB的常
量内存，并且对常量内存采取了不同于标准全局内存的处理方式。在某些情况中，用常量
内存来替换全局内存能有效地减少内存带宽。 在某些情况下，使用常量内存将提升应用程
序的性能 */
__constant__ Sphere dev_spheres[20]; // 常量内存, = sphere_num

/* __global__: 函数类型限定符;在设备上运行;在主机端调用,计算能力3.2及以上可以在
设备端调用;声明的函数的返回值必须是void类型;对此类型函数的调用是异步的,即在
设备完全完成它的运行之前就返回了;对此类型函数的调用必须指定执行配置,即用于在
设备上执行函数时的grid和block的维度,以及相关的流(即插入<<<   >>>运算符);
a kernel,表示此函数为内核函数(运行在GPU上的CUDA并行计算函数称为kernel(内核函
数),内核函数必须通过__global__函数类型限定符定义); */
__global__ static void ray_tracking(unsigned char* ptr_image, Sphere* ptr_sphere, int width, int height, int sphere_num)
{
	/* gridDim: 内置变量,用于描述线程网格的维度,对于所有线程块来说,这个
	变量是一个常数,用来保存线程格每一维的大小,即每个线程格中线程块的数量.
	一个grid为三维,为dim3类型；
	blockDim: 内置变量,用于说明每个block的维度与尺寸.为dim3类型,包含
	了block在三个维度上的尺寸信息;对于所有线程块来说,这个变量是一个常数,
	保存的是线程块中每一维的线程数量;
	blockIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程块的索引;用
	于说明当前thread所在的block在整个grid中的位置,blockIdx.x取值范围是
	[0,gridDim.x-1],blockIdx.y取值范围是[0, gridDim.y-1].为uint3类型,
	包含了一个block在grid中各个维度上的索引信息;
	threadIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程索引;用于
	说明当前thread在block中的位置;如果线程是一维的可获取threadIdx.x,如果
	是二维的还可获取threadIdx.y,如果是三维的还可获取threadIdx.z;为uint3类
	型,包含了一个thread在block中各个维度的索引信息 */
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	float ox{ (x - width / 2.f) };
	float oy{ (y - height / 2.f) };

	float r{ 0 }, g{ 0 }, b{ 0 };
	float maxz{ -INF };

	for (int i = 0; i < sphere_num; ++i) {
		float n;
		float t = ptr_sphere[i].hit(ox, oy, &n);
		if (t > maxz) {
			float fscale = n;
			r = ptr_sphere[i].r * fscale;
			g = ptr_sphere[i].g * fscale;
			b = ptr_sphere[i].b * fscale;
			maxz = t;
		}
	}

	ptr_image[offset * 4 + 0] = static_cast<unsigned char>(r * 255);
	ptr_image[offset * 4 + 1] = static_cast<unsigned char>(g * 255);
	ptr_image[offset * 4 + 2] = static_cast<unsigned char>(b * 255);
	ptr_image[offset * 4 + 3] = 255;
}

__global__ static void ray_tracking(unsigned char* ptr_image, int width, int height, int sphere_num)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	float ox{ (x - width / 2.f) };
	float oy{ (y - height / 2.f) };

	float r{ 0 }, g{ 0 }, b{ 0 };
	float maxz{ -INF };

	for (int i = 0; i < sphere_num; ++i) {
		float n;
		float t = dev_spheres[i].hit(ox, oy, &n);
		if (t > maxz) {
			float fscale = n;
			r = dev_spheres[i].r * fscale;
			g = dev_spheres[i].g * fscale;
			b = dev_spheres[i].b * fscale;
			maxz = t;
		}
	}

	ptr_image[offset * 4 + 0] = static_cast<unsigned char>(r * 255);
	ptr_image[offset * 4 + 1] = static_cast<unsigned char>(g * 255);
	ptr_image[offset * 4 + 2] = static_cast<unsigned char>(b * 255);
	ptr_image[offset * 4 + 3] = 255;
}

int ray_tracking_gpu(const float* a, const float* b, const float* c, int sphere_num, unsigned char* ptr, int width, int height, float* elapsed_time)
{
	/* hipEvent_t: CUDA event types,结构体类型, CUDA事件,用于测量GPU在某
	个任务上花费的时间,CUDA中的事件本质上是一个GPU时间戳,由于CUDA事件是在
	GPU上实现的,因此它们不适于对同时包含设备代码和主机代码的混合代码计时 */
	hipEvent_t start, stop;
	// hipEventCreate: 创建一个事件对象,异步启动
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: 记录一个事件,异步启动,start记录起始时间
	hipEventRecord(start, 0);

	const size_t length{ width * height * 4 * sizeof(unsigned char) };
	unsigned char* dev_image{ nullptr };

	std::unique_ptr<Sphere[]> spheres(new Sphere[sphere_num]);
	for (int i = 0, t = 0; i < sphere_num; ++i, t += 3) {
		spheres[i].r = a[t];
		spheres[i].g = a[t + 1];
		spheres[i].b = a[t + 2];
		spheres[i].x = b[t];
		spheres[i].y = b[t + 1];
		spheres[i].z = b[t + 2];
		spheres[i].radius = c[i];
	}

	// hipMalloc: 在设备端分配内存
	hipMalloc(&dev_image, length);

	// method1: 没有使用常量内存
	//Sphere* dev_spheres{ nullptr };
	//hipMalloc(&dev_spheres, sizeof(Sphere) * sphere_num);
	/* hipMemcpy: 在主机端和设备端拷贝数据,此函数第四个参数仅能是下面之一:
	(1). hipMemcpyHostToHost: 拷贝数据从主机端到主机端
	(2). hipMemcpyHostToDevice: 拷贝数据从主机端到设备端
	(3). hipMemcpyDeviceToHost: 拷贝数据从设备端到主机端
	(4). hipMemcpyDeviceToDevice: 拷贝数据从设备端到设备端
	(5). hipMemcpyDefault: 从指针值自动推断拷贝数据方向,需要支持
	统一虚拟寻址(CUDA6.0及以上版本)
	hipMemcpy函数对于主机是同步的 */
	//hipMemcpy(dev_spheres, spheres.get(), sizeof(Sphere) * sphere_num, hipMemcpyHostToDevice);

	// method2: 使用常量内存
	/* hipMemcpyToSymbol: hipMemcpyToSymbol和hipMemcpy参数为
	hipMemcpyHostToDevice时的唯一差异在于hipMemcpyToSymbol会复制到常量内
	存，而hipMemcpy会复制到全局内存 */
	hipMemcpyToSymbol(HIP_SYMBOL(dev_spheres), spheres.get(), sizeof(Sphere)* sphere_num);

	const int threads_block{ 16 };
	/* dim3: 基于uint3定义的内置矢量类型，相当于由3个unsigned int类型组成的
	结构体，可表示一个三维数组，在定义dim3类型变量时，凡是没有赋值的元素都
	会被赋予默认值1 */
	dim3 blocks(width / threads_block, height / threads_block);
	dim3 threads(threads_block, threads_block);

	/* <<< >>>: 为CUDA引入的运算符,指定线程网格和线程块维度等,传递执行参
	数给CUDA编译器和运行时系统,用于说明内核函数中的线程数量,以及线程是如何
	组织的;尖括号中这些参数并不是传递给设备代码的参数,而是告诉运行时如何
	启动设备代码,传递给设备代码本身的参数是放在圆括号中传递的,就像标准的函
	数调用一样;不同计算能力的设备对线程的总数和组织方式有不同的约束;必须
	先为kernel中用到的数组或变量分配好足够的空间,再调用kernel函数,否则在
	GPU计算时会发生错误,例如越界等;
	使用运行时API时,需要在调用的内核函数名与参数列表直接以<<<Dg,Db,Ns,S>>>
	的形式设置执行配置,其中：Dg是一个dim3型变量,用于设置grid的维度和各个
	维度上的尺寸.设置好Dg后,grid中将有Dg.x*Dg.y*Dg.z个block;Db是
	一个dim3型变量,用于设置block的维度和各个维度上的尺寸.设置好Db后,每个
	block中将有Db.x*Db.y*Db.z个thread;Ns是一个size_t型变量,指定各块为此调
	用动态分配的共享存储器大小,这些动态分配的存储器可供声明为外部数组
	(extern __shared__)的其他任何变量使用;Ns是一个可选参数,默认值为0;S为
	hipStream_t类型,用于设置与内核函数关联的流.S是一个可选参数,默认值0. */
	//ray_tracking << <blocks, threads >> >(dev_image, dev_spheres, width, height, sphere_num); // method1, 不使用常量内存
	ray_tracking << <blocks, threads >> >(dev_image, width, height, sphere_num); // method2, 使用常量内存

	hipMemcpy(ptr, dev_image, length, hipMemcpyDeviceToHost);

	// hipFree: 释放设备上由hipMalloc函数分配的内存
	hipFree(dev_image);
	//hipFree(dev_spheres); // 使用method1时需要释放, 如果使用常量内存即method2则不需要释放

	// hipEventRecord: 记录一个事件,异步启动,stop记录结束时间
	hipEventRecord(stop, 0);
	// hipEventSynchronize: 事件同步,等待一个事件完成,异步启动
	hipEventSynchronize(stop);
	// cudaEventElapseTime: 计算两个事件之间经历的时间,单位为毫秒,异步启动
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: 销毁事件对象,异步启动
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
